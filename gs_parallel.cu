#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <sys/time.h>

#define MAX_ITER 1000
#define MAX 100 // maximum value of the matrix element
#define ERROR 0.000001

// Allocate 2D matrix

void init_bound_cond(float ***u, int m, int n)
{
    int i, j;
    *u = (float **)malloc(sizeof(float *) * (m));
    for (i = 0; i < m; i++)
    {
        (*u)[i] = (float *)malloc(sizeof(float) * (n));
    }

    // Initialize boundary conditon
    for (j = 1; j < n - 1; j++)
    {
        (*u)[0][j] = 50;
        (*u)[m - 1][j] = 300;
    }
    for (i = 1; i < m - 1; i++)
    {
        (*u)[i][0] = 75;
        (*u)[i][n - 1] = 100;
    }
}

void print_mat(float **a, int m, int n)
{
    int i, j;
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", a[i][j]);
        }
        printf("\n");
    }
}

// solver

__global__ void solve(float **matdi, float **matdo, int n, int m, float *diff)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    

    // printf("%d, %d\n", i, j);
    if ((i > 0) && (j > 0) && (i < (n - 1)) && (j < (m - 1)))
    {
        (matdo)[i][j] = 0.25 * ((matdi)[i][j - 1] + (matdi)[i - 1][j] + (matdi)[i][j + 1] + (matdi)[i + 1][j]);
        atomicAdd(diff, abs((matdo)[i][j] - (matdi)[i][j]));
    }
}

int main(int argc, char *argv[])
{

    int n, m, cnt_iter = 0;
    float **a, **adi, **ado, *d_diff;
    
    n = 6; m = 5;
    init_bound_cond(&a, n, m);

    // float **temp;
    float **temi, **temo;
    temi = new float *[n];
    temo = new float *[n];


    print_mat(a, n, m);
 

    hipMalloc(&adi, n * sizeof(float *));
    hipMalloc(&ado, n * sizeof(float *));
    for (int i = 0; i < n; i++)
    {
        hipMalloc(&(temi[i]), m * sizeof(float));
        hipMalloc(&(temo[i]), m * sizeof(float));
        hipMemcpy(temi[i], a[i], m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(temo[i], a[i], m * sizeof(float), hipMemcpyHostToDevice);
    }
    hipMemcpy(adi, temi, n * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(ado, temo, n * sizeof(float *), hipMemcpyHostToDevice);
    float h_diff = n * m;
    dim3 DimBlock(32, 8);
    dim3 DimGrid((n + DimBlock.x - 1) / DimBlock.x, (m + DimBlock.y - 1) / DimBlock.y);

    // So luong grid tren moi chieu sao cho, grid_dim_x * block_dim_x >= n; tuong tu voi m

    hipMalloc(&d_diff, sizeof(float));
    hipMemset(d_diff, 0, sizeof(float));
    while ((cnt_iter < MAX_ITER) && ((h_diff / (n * m)) > ERROR))
    {
        solve<<<DimGrid, DimBlock>>>(adi, ado, n, m, d_diff);
        hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost);
        hipMemset(d_diff, 0, sizeof(float));
        cnt_iter++;
        float **adt = adi; // Swap current result voi cai cu
        adi = ado;
        ado = adt;
    }

    printf("The algorithm converges after %d with difference = %f\n", cnt_iter, h_diff / (n * m));
    for (int i = 0; i < n; i++)
    {
        hipMemcpy(a[i], temi[i], m * sizeof(float), hipMemcpyDeviceToHost);
    }

    print_mat(a, n, m);

    return 0;
}